#include "hip/hip_runtime.h"
// nvcc --shared --compiler-options -fPIC -o libjnet.so jnet.cu -lcublas
// TODO:
// train
// update: check L2, momentum, nesterov
// update: implement maxnorm, L1
// dropout
// compare with caffe, matlab

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include "jnet.h"
#define BLK 128
#define THR 128

static inline float *gpuArray(int nfloats);
static inline float *gpuCopy(int nfloats, float *cpuArray);
static inline float *gpuFill(int nfloats, float val);
static inline void fforw(Layer l);
static inline float *initforw(Layer l, float *x, int xcols);
static inline void fback(Layer l);
static inline float *initback(Layer l, float *dy, int dx);
static inline void initupdate(Layer l);
__global__ void _reluforw(int n, float *y);
__global__ void _reluback(int n, float *y, float *dy);
__global__ void _softback(int nrows, int ncols, float *y, float *dy);
__global__ void _l1reg(int n, float l1, float *w, float *dw);
__global__ void _adagrad(int n, float *dw2, float *dw);
__global__ void _fill(int n, float val, float *x);

#define CUDA(_s) assert((_s) == hipSuccess)
#define CUBLAS(_s) assert(((_s), cublasGetError())==HIPBLAS_STATUS_SUCCESS)

#define gpuFree(x) CUDA(hipFree(x))
#define gpuGetMatrix(rows,cols,from,to) CUBLAS(hipblasGetMatrix(rows,cols,sizeof(float),from,rows,to,rows))
#define gpuSetMatrix(rows,cols,from,to) CUBLAS(hipblasSetMatrix(rows,cols,sizeof(float),from,rows,to,rows))

static inline float *gpuArray(int nfloats) {
  float *gptr;
  CUDA(hipMalloc((void **) &gptr, nfloats * sizeof(float)));
  return gptr;
}

static inline float *gpuCopy(int nfloats, float *cpuArray) {
  float *gptr = gpuArray(nfloats);
  CUDA(hipMemcpy(gptr, cpuArray, nfloats * sizeof(float), hipMemcpyHostToDevice));
  return gptr;
}

static inline float *gpuFill(int nfloats, float val) {
  float *gptr = gpuArray(nfloats);
  _fill<<<BLK,THR>>>(nfloats, val, gptr);
  return gptr;
}


Layer layer(LayerType type, int wrows, int wcols, float *w, float *b) {
  static int init = 0;
  if (!init) { CUBLAS(cublasInit()); init=1; }
  Layer l = (Layer) calloc(1, sizeof(struct LayerS));
  assert(l != NULL);
  l->type = type;
  l->wrows = wrows;
  l->wcols = wcols;
  assert(w != NULL);
  l->w = gpuCopy(wrows*wcols, w);
  if (b != NULL) l->b = gpuCopy(wrows, b);
  return(l);
}

void lfree(Layer l) {
  gpuFree(l->w);
  gpuFree(l->b);
  //gpuFree(l->x);   // taken as input, not alloced
  gpuFree(l->y);   
  gpuFree(l->xones);
  //gpuFree(l->xmask);

  gpuFree(l->dw);
  gpuFree(l->db);
  gpuFree(l->dx);
  //gpuFree(l->dy);  // taken as input, not alloced

  gpuFree(l->dw1);
  gpuFree(l->dw2);
  gpuFree(l->db1);
  gpuFree(l->db2);
  free(l);
}

void set_adagrad(Layer l, int i) { l->adagrad = i; }
void set_nesterov(Layer l, int i) { l->nesterov = i; }
void set_learningRate(Layer l, float f) { l->learningRate = f; }
void set_momentum(Layer l, float f) { l->momentum = f; }
void set_dropout(Layer l, float f) { l->dropout = f; }
void set_maxnorm(Layer l, float f) { l->maxnorm = f; }
void set_L1(Layer l, float f) { l->L1 = f; }
void set_L2(Layer l, float f) { l->L2 = f; }

int lsize(Layer l, int i) { 
  return (i==1 ? l->wrows : i==2 ? l->wcols : 1);
}

Layer relu(int wrows, int wcols, float *w, float *b) {
  return layer(RELU, wrows, wcols, w, b);
}

Layer soft(int wrows, int wcols, float *w, float *b) {
  return layer(SOFT, wrows, wcols, w, b);
}

void forward(Layer *net, float *x, float *y, int nlayer, int xcols, int batch) {
  int xrows = net[0]->wcols;
  int yrows = net[nlayer-1]->wrows;
  float *xgpu = gpuArray(xrows * batch);
  for (int b = 0; b < xcols; b += batch) {
    if (b + batch > xcols) batch = xcols - b;
    gpuSetMatrix(xrows, batch, &x[b*xrows], xgpu);
    float *gptr = xgpu;
    for (int l = 0; l < nlayer; l++)
      gptr = lforw(net[l], gptr, batch);
    gpuGetMatrix(yrows, batch, gptr, &y[b*yrows]);
  }
  gpuFree(xgpu);
}

void forwback(Layer *net, float *x, float *y, int nlayer, int xcols, int batch) {
  int xrows = net[0]->wcols;
  int yrows = net[nlayer-1]->wrows;
  float *xgpu = gpuArray(xrows * batch);
  float *ygpu = gpuArray(yrows * batch);
  for (int b = 0; b < xcols; b += batch) {
    if (b + batch > xcols) batch = xcols - b;
    gpuSetMatrix(xrows, batch, &x[b*xrows], xgpu);
    float *gptr = xgpu;
    for (int l = 0; l < nlayer; l++)
      gptr = lforw(net[l], gptr, batch);
    gpuSetMatrix(yrows, batch, &y[b*yrows], ygpu);
    gptr = ygpu;
    for (int l = nlayer - 1; l >= 0; l--)
      gptr = lback(net[l], gptr, (l>0));
  }
  gpuFree(xgpu);
  gpuFree(ygpu);
}


float *lforw(Layer l, float *x, int xcols) {
  // We assume x is already a device pointer.
  // Otherwise we'd have to do unnecessary copying between layers.
  // We assume xrows == l->wcols and x is column-major.  
  l->x = initforw(l, x, xcols);

  // y = w * x
  // gemm(opA,opB,m,n,k,α,A(m,k),lda=m,B(k,n),ldb=k,β,C(m,n),ldc=m): C = α op(A) op(B) + β C
  CUBLAS(hipblasSgemm('N', 'N', l->wrows, l->xcols, l->wcols, 1.0, l->w, l->wrows, l->x, l->wcols, 0.0, l->y, l->wrows));

  if (l->b != NULL) {
    // y = y + b  with singleton expansion
    // ger(m,n,α,x(m),incx=1,y(n),incy=1,A(m,n),lda=m): A = α x y' + A
    CUBLAS(hipblasSger(l->wrows, l->xcols, 1.0, l->b, 1, l->xones, 1, l->y, l->wrows));
  }
  // y = f(y) where f is relu, sigm etc.
  fforw(l);
  return l->y;
}

static inline void fforw(Layer l) {
  switch(l->type) {
  case RELU:
    _reluforw<<<BLK,THR>>>(l->wrows * l->xcols, l->y);
    CUDA(hipGetLastError());
    break;
  }
}

static inline float *initforw(Layer l, float *x, int xcols) {
  // Alloc/realloc l->y and l->xones if necessary and update l->xcols
  int yrows = l->wrows;
  int ycols = xcols;
  if ((l->y == NULL) || (l->xcols != xcols)) {
    gpuFree(l->y);
    l->y = gpuArray(yrows * ycols);
  }
  if ((l->b != NULL) && ((l->xones == NULL) || l->xcols != xcols)) {
    gpuFree(l->xones);
    l->xones = gpuFill(xcols, 1.0);
  }
  if ((l->dx != NULL) && (l->xcols != xcols)) {
    gpuFree(l->dx);
    l->dx = NULL;	/* to be reallocated */
  }
  l->xcols = xcols;
  return x;
}

float *lback(Layer l, float *dy, int dx) {
  // We assume dy is already a device pointer.
  // Otherwise we'd have to do unnecessary copying between layers.
  // We assume dy has the same size as l.y: (wrows,xcols)
  l->dy = initback(l, dy, dx);
  
  // dy = fback(dy) where fback is the derivative of fforw
  fback(l);

  // dw = dy * x'
  // gemm(opA,opB,m,n,k,α,A(m,k),lda=m,B(k,n),ldb=k,β,C(m,n),ldc=m): C = α op(A) op(B) + β C
  // m = wrows; n = wcols; k = xcols
  CUBLAS(hipblasSgemm('N', 'T', l->wrows, l->wcols, l->xcols, 1.0, l->dy, l->wrows, l->x, l->wcols, 0.0, l->dw, l->wrows));

  if (l->b != NULL) {
    // db = sum(dy,2) = dy * ones
    // gemv(op,m,n,α,A(m,n),lda=m,x(n),incx=1,β,y(m),incy=1): y = α op(A) x + β y
    CUBLAS(hipblasSgemv('N', l->wrows, l->xcols, 1.0, l->dy, l->wrows, l->xones, 1, 0.0, l->db, 1));
  }
  if (dx) { // dx is optional because it is expensive and unnecessary for input layer
    // dx=w' * dy
    // gemm(opA,opB,m,n,k,α,A(m,k),lda=m,B(k,n),ldb=k,β,C(m,n),ldc=m): C = α op(A) op(B) + β C
    // m = wcols, n = xcols, k = wrows
    CUBLAS(hipblasSgemm('T', 'N', l->wcols, l->xcols, l->wrows, 1.0, l->w, l->wrows, l->dy, l->wrows, 0.0, l->dx, l->wcols));
  }
  return l->dx;
}

static inline void fback(Layer l) {
  switch(l->type) {
  case RELU:
    _reluback<<<BLK,THR>>>(l->wrows * l->xcols, l->y, l->dy);
    CUDA(hipGetLastError());
    break;
  case SOFT:
    _softback<<<BLK,THR>>>(l->wrows, l->xcols, l->y, l->dy);
    CUDA(hipGetLastError());
    break;
  }
}

static inline float *initback(Layer l, float *dy, int dx) {
  if (l->dw == NULL) l->dw = gpuArray(l->wrows * l->wcols);
  if ((l->b != NULL) && (l->db == NULL)) l->db = gpuArray(l->wrows);
  if (dx && (l->dx == NULL)) l->dx = gpuArray(l->wcols * l->xcols);
  return dy;
}

void update(Layer l) {
  initupdate(l);
  if (l->learningRate == 0) return;
  int nw = l->wcols * l->wrows;
  int nb = (l->b == NULL ? 0 : l->wrows);

  if (l->L1 != 0) {
    /* L1 regularization:
       J(w,b) = Jerr + L1 Σ|wi|
       ∂J/∂wi = ∂Jerr/∂wi + L1 sign(wi)
       dw contains ∂Jerr/∂wi after lback
       we want: dw += L1 * sign(w)
       axpy(n,α,x(n),incx=1,y(n),incy=1): y = α x + y
    */
    _l1reg<<<BLK,THR>>>(nw, l->L1, l->w, l->dw);
    CUDA(hipGetLastError());
  }

  if (l->L2 != 0) { 
    /* L2 regularization:
       J(w,b) = Jerr + (L2/2)|w|^2
       ∂J/∂wi = ∂Jerr/∂wi + L2 wi 
       dw contains ∂Jerr/∂wi after lback
       we want: dw += L2 * w
       axpy(n,α,x(n),incx=1,y(n),incy=1): y = α x + y
    */
    hipblasSaxpy(nw, l->L2, l->w, 1, l->dw, 1);
  }
  if (l->adagrad) {
    /* ADAGRAD:
       dw2 += dw.*dw 
       dw /= (epsilon + sqrt(dw2))
       and similarly for db.
    */
    _adagrad<<<BLK,THR>>>(nw, l->dw2, l->dw);
    if (nb) { _adagrad<<<BLK,THR>>>(nb, l->db2, l->db); }
    CUDA(hipGetLastError());
  }
  if (l->learningRate != 1) {
    /* LearningRate:
       Scale dw and db with the learning rate.
       dw,db *= learningRate
       scal(n,α,x(n),incx=1): x = α x
    */
    hipblasSscal(nw, l->learningRate, l->dw, 1);
    if (nb) hipblasSscal(nb, l->learningRate, l->db, 1);
  }
  if (l->momentum != 0) {  
    /* Momentum:
       why do we apply it here?
       do we apply it to db?
       check the following:
       dw1 = momentum * dw1 + dw
       dw = dw1   :without nesterov
       dw = momentum * dw1 + dw   :with nesterov
    */
    assert(1==0);	/* need to check first */
    hipblasSscal(nw, l->momentum, l->dw1, 1);
    hipblasSaxpy(nw, 1.0, l->dw, 1, l->dw1, 1);
    if (l->nesterov) {
      hipblasSaxpy(nw, l->momentum, l->dw1, 1, l->dw, 1);
    } else {
      hipblasScopy(nw, l->dw1, 1, l->dw, 1);
    }
  }
  /* Finally apply gradient descent: w -= dw, b -= db */
  hipblasSaxpy(nw, -1.0, l->dw, 1, l->w, 1);
  if (nb) hipblasSaxpy(nb, -1.0, l->db, 1, l->b, 1);

  if (l->maxnorm != 0) {
    /* MaxNorm:

     */
    assert(1 == 0);  // TBD
  }
}

static inline void initupdate(Layer l) {
  if (l->adagrad) {
    if (l->dw2 == NULL) l->dw2 = gpuFill(l->wrows * l->wcols, 0.0);
    if ((l->b != NULL) && (l->db2 == NULL)) l->db2 = gpuFill(l->wrows, 0.0);
  }
  if (l->momentum != 0) {
    if (l->dw1 == NULL) l->dw1 = gpuFill(l->wrows * l->wcols, 0.0);
    if ((l->b != NULL) && (l->db1 == NULL)) l->db1 = gpuFill(l->wrows, 0.0);
  }
}


__global__ void _fill(int n, float val, float *x) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    x[i] = val;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _adagrad(int n, float *dw2, float *dw) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    dw2[i] += dw[i] * dw[i];
    dw[i] /= (1e-8 + sqrt(dw2[i]));
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _l1reg(int n, float l1, float *w, float *dw) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    dw[i] += (w[i] >= 0 ? l1 : -l1);
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _reluforw(int n, float *y) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (y[i] < 0) y[i] = 0;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _reluback(int n, float *y, float *dy) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (y[i] <= 0) dy[i] = 0;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _softback(int nrows, int ncols, float *y, float *dy) {
  float y0, sum;
  int i0, i1;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  while (col < ncols) {
    i0 = col * nrows;
    i1 = i0  + nrows;
    y0 = -INFINITY;
    //y0 = y[i0];
    for (int i=i0; i<i1; i++) {
      if (y[i] > y0) {
	y0 = y[i];
      }
    }
    sum = 0;
    for (int i=i0; i<i1; i++) {
      y[i] = exp(y[i]-y0);
      sum += y[i];
    }
    for (int i=i0; i<i1; i++) {
      y[i] /= sum;
      dy[i] = (y[i] - dy[i]) / ncols;
    }
    col += blockDim.x * gridDim.x;
  }
}

