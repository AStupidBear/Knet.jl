#include "hip/hip_runtime.h"
// nvcc --shared --compiler-options -fPIC -o libjnet.so jnet.cu -lcublas

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "jnet.h"
#define BLK 128
#define THR 128

typedef enum { NOOP, RELU, SOFT } LayerType;

typedef struct LayerS {
  LayerType type; // type of activation function	
  float *w;	// weight matrix (wrows,wcols)
  float *b;	// bias vector (wrows)

  float *dw;	// gradient wrt weight matrix
  float *dw1;	// moving average of gradients for momentum
  float *dw2;	// sum of squared gradients for adagrad

  float *db;	// gradient wrt bias vector
  float *db1;	// moving average of gradients for momentum
  float *db2;	// sum of squared gradients for adagrad

  float *x;	// last input (wcols,xcols)
  float *y;	// last output (wrows,xcols)
  float *dx;	// gradient wrt input
  float *dy;	// gradient wrt output
  float *xmask;	// input mask for dropout
  float *xones;	// vector of ones for bias calculation (xcols)

  int wrows;
  int wcols;
  int xcols;
} *Layer;


Layer layer(LayerType type, int wrows, int wcols, float *w, float *b);
float *forw(Layer l, float *x, int xcols);
float *initforw(Layer l, float *x, int xcols);

__global__ void _fill(float *y, int n, float val);
__global__ void _reluforw(float *y, int n);
__global__ void _reluback(float *dy, float *y, int n);
__global__ void _softback(float *dy, float *y, int nrows, int ncols);


#define CUDA(s) {\
    hipError_t err = (s);\
    if (err != hipSuccess) {\
      fprintf(stderr, "cuda error: %s\n", hipGetErrorString(err));\
      exit(EXIT_FAILURE);\
    }\
  }


#define NOTNULL(s) {\
    if ((s) == NULL) {\
      fprintf(stderr, "Unexpected null value\n");\
      exit(EXIT_FAILURE);\
    }\
  }


Layer layer(LayerType type, int wrows, int wcols, float *w, float *b) {
  Layer l;
  NOTNULL(w);
  NOTNULL(l = (Layer) calloc(1, sizeof(struct LayerS)));
  l = (Layer) calloc(1, sizeof(struct LayerS));
  l->type = type;
  l->wrows = wrows;
  l->wcols = wcols;
  int wsize = wrows*wcols*sizeof(float);
  CUDA(hipMalloc((void **) &l->w, wsize));
  CUDA(hipMemcpy(l->w, w, wsize, hipMemcpyHostToDevice));
  if (b != NULL) {
    int bsize = wrows*sizeof(float);
    CUDA(hipMalloc((void **) &l->b, bsize));
    CUDA(hipMemcpy(l->b, b, bsize, hipMemcpyHostToDevice));
  }
  return(l);
}

extern "C" void lfree(Layer l) {
  CUDA(hipFree(l->w));
  if (l->b != NULL) CUDA(hipFree(l->b));
  free(l);
}

extern "C" int lsize(Layer l, int i) { 
  switch(i) {
  case 1: return l->wrows;
  case 2: return l->wcols;
  default: fprintf(stderr, "size argument must be 1 or 2\n");
    exit(EXIT_FAILURE);
  }
}

extern "C" Layer relu(int wrows, int wcols, float *w, float *b) {
  return layer(RELU, wrows, wcols, w, b);
}

extern "C" Layer soft(int wrows, int wcols, float *w, float *b) {
  return layer(SOFT, wrows, wcols, w, b);
}

extern "C" void forward(Layer *net, float *x, float *y, int nlayer, int xcols, int batch) {
  float *xgpu;
  int xrows = net[0]->wcols;
  int yrows = net[nlayer-1]->wrows;
  int xsize = xrows * batch * sizeof(float);
  int ysize = yrows * batch * sizeof(float);
  CUDA(hipMalloc((void **) &xgpu, xsize));
  for (int b = 0; b < xcols; b += batch) {
    if (b + batch > xcols) {
      batch = xcols - b;
      xsize = xrows * batch * sizeof(float);
      ysize = yrows * batch * sizeof(float);
    }
    CUDA(hipMemcpy(xgpu, &x[b * xrows], xsize, hipMemcpyHostToDevice));
    float *ygpu = xgpu;
    for (int l = 0; l < nlayer; l++) {
      ygpu = forw(net[l], ygpu, batch);
    }
    CUDA(hipMemcpy(&y[b * yrows], ygpu, ysize, hipMemcpyDeviceToHost));
  }
  CUDA(hipFree(xgpu));
}

float *forw(Layer l, float *x, int xcols) {
  // We assume x is already a device pointer.
  // Otherwise we'd have to do unnecessary copying between layers.
  // We assume xrows == l->wcols and x is column-major.  
  l->x = initforw(l, x, xcols);

  // l->y = l->w * l->x
  // gemm(opA,opB,m,n,k,α,A(m,k),lda=m,B(k,n),ldb=k,β,C(m,n),ldc=m): C = α op(A) op(B) + β C
  hipblasSgemm('N', 'N', l->wrows, l->xcols, l->wcols, 1.0, l->w, l->wrows, l->x, l->wcols, 0.0, l->y, l->wrows);

  if (l->b != NULL) {
    // l->y = l->y + l->b with singleton expansion
    // ger(m,n,α,x(m),incx=1,y(n),incy=1,A(m,n),lda=m): A = α x y' + A
    hipblasSger(l->wrows, l->xcols, 1.0, l->b, 1, l->xones, 1, l->y, l->wrows);
  }
  // l->y = fforw(l->y)
  switch(l->type) {
  case RELU:
    _reluforw<<<BLK,THR>>>(l->y, l->wrows * l->xcols);
    CUDA(hipGetLastError());
    break;
  }
  return(l->y);
}

float *initforw(Layer l, float *x, int xcols) {
  // Alloc/realloc l->y and l->xones if necessary
  // Update l->xcols
  int yrows = l->wrows;
  int ycols = xcols;
  if ((l->y == NULL) || (l->xcols != xcols)) {
    CUDA(hipFree(l->y));
    CUDA(hipMalloc((void **) &l->y, yrows * ycols * sizeof(float)));
  }
  if ((l->b != NULL) && ((l->xones == NULL) || l->xcols != xcols)) {
    CUDA(hipFree(l->xones));
    CUDA(hipMalloc((void **) &l->xones, xcols * sizeof(float)));
    _fill<<<BLK,THR>>>(l->xones, xcols, 1.0);
    CUDA(hipGetLastError());
  }
  l->xcols = xcols;
  return x;
}

__global__ void _fill(float *y, int n, float val)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    y[i] = val;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _reluforw(float *y, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (y[i] < 0) y[i] = 0;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _reluback(float *dy, float *y, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (y[i] <= 0) dy[i] = 0;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _softback(float *dy, float *y, int nrows, int ncols)
{
  float y0, sum;
  int i0, i1;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  while (col < ncols) {
    i0 = col * nrows;
    i1 = i0  + nrows;
    y0 = -INFINITY;
    //y0 = y[i0];
    for (int i=i0; i<i1; i++) {
      if (y[i] > y0) {
	y0 = y[i];
      }
    }
    sum = 0;
    for (int i=i0; i<i1; i++) {
      y[i] = exp(y[i]-y0);
      sum += y[i];
    }
    for (int i=i0; i<i1; i++) {
      y[i] /= sum;
      dy[i] = (y[i] - dy[i]) / ncols;
    }
    col += blockDim.x * gridDim.x;
  }
}

