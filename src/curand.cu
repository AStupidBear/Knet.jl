#include "kunet.h"
#include <hiprand.h>
hiprandGenerator_t RNG;
#define CURAND(_s) {							\
    if (RNG==NULL) assert(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT)==HIPRAND_STATUS_SUCCESS); \
    assert((_s) == HIPRAND_STATUS_SUCCESS);				\
  }

extern "C" {

  void *gpurng() { return RNG; }

  void gpuseed(unsigned long long seed) {
    // need to regenerate RNG for the seed to take effect
    assert(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT)==HIPRAND_STATUS_SUCCESS);
    assert(hiprandSetPseudoRandomGeneratorSeed(RNG, seed)==HIPRAND_STATUS_SUCCESS);
  }

  void rand32(float* x, int n) CURAND(hiprandGenerateUniform(RNG, x, n));
  void rand64(double *x, int n) CURAND(hiprandGenerateUniformDouble(RNG, x, n));

  void randn32(float *x, int n, float mean, float std) {
    // this requires n to be even, so we'll fix it here
    if (n%2 == 0) {
      CURAND(hiprandGenerateNormal(RNG, x, n, mean, std));
    } else {
      CURAND(hiprandGenerateNormal(RNG, x, n, mean, std));
      CURAND(hiprandGenerateNormal(RNG, x, n, mean, std));
    }
  }
}
