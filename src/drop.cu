#include "hip/hip_runtime.h"
#include "kunet.h"

__global__ void _drop(int n, float *x, float *xmask, float dropout, float scale) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (xmask[i] < dropout) x[i] = 0;
    else x[i] *= scale;
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
void drop(int n, float *x, float *xmask, float dropout, float scale) KCALL(_drop,n,x,xmask,dropout,scale);
}
