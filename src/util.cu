#include "kunet.h"
#include <hiprand.h>
#define CURAND(_s) assert((_s) == HIPRAND_STATUS_SUCCESS)

/* TODO: These should be directly called from julia. */

static hiprandGenerator_t RNG;

extern "C" {

void gpuseed(unsigned long long seed) {
  CURAND(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND(hiprandSetPseudoRandomGeneratorSeed(RNG, seed));
}

void randfill32(int n, float *x) {
  if (RNG == NULL) CURAND(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND(hiprandGenerateUniform(RNG, x, n));
}

void randfill64(int n, double *x) {
  if (RNG == NULL) CURAND(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND(hiprandGenerateUniformDouble(RNG, x, n));
}

}
