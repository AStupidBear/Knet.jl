#include "kunet.h"

/* TODO: These should be directly called from julia. */

hiprandGenerator_t RNG;

extern "C" {

  void gpuseed(unsigned long long seed) {
    assert(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT)==HIPRAND_STATUS_SUCCESS);
    assert(hiprandSetPseudoRandomGeneratorSeed(RNG, seed)==HIPRAND_STATUS_SUCCESS);
  }

  void randfill32(int n, float *x) CURAND(hiprandGenerateUniform(RNG, x, n));

  void randfill64(int n, double *x) CURAND(hiprandGenerateUniformDouble(RNG, x, n));

}
