#include "hip/hip_runtime.h"
#include "kunet.h"

__global__ void _add1(int n, float val, float *x) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    x[i] += val;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _badd(int nrows, int ncols, float *y, float *b) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int n = nrows * ncols;
  while (i < n) {
    y[i] += b[i % nrows];
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void add1(int n, float val, float *x) KCALL(_add1,n,val,x);
  void badd(int nrows, int ncols, float *y, float *b) KCALL(_badd,nrows,ncols,y,b);
}
