#include "kunet.h"

/* TODO: These should be directly called from julia. */

hiprandGenerator_t RNG;

extern "C" {

  void gpuseed(unsigned long long seed) {
    assert(hiprandCreateGenerator(&RNG, HIPRAND_RNG_PSEUDO_DEFAULT)==HIPRAND_STATUS_SUCCESS);
    assert(hiprandSetPseudoRandomGeneratorSeed(RNG, seed)==HIPRAND_STATUS_SUCCESS);
  }

  void randfill32(int n, float *x) CURAND(hiprandGenerateUniform(RNG, x, n));
  void randfill64(int n, double *x) CURAND(hiprandGenerateUniformDouble(RNG, x, n));
  void initgaussian32(float *x, int n, float mean, float std) CURAND(hiprandGenerateNormal(RNG, x, n, mean, std));
  void initgaussian64(double *x, int n, double mean, double std) CURAND(hiprandGenerateNormalDouble(RNG, x, n, mean, std));
  void *rng() { return RNG; }
}
