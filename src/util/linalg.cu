#include "hip/hip_runtime.h"
#include "../kunet.h"

/*
  The following functions multiply two sparse matrices into a dense matrix.
  The sparse matrices are in 1-based csc format.
  Ast_mul_Bs uses the transpose of the first arg and a simpler algorithm.
  x(nd,nx) s(nd,ns) -> k(nx,ns)
  As_mul_Bs uses the fast algorithm from the Julia sparse code.
  x(nx,nd) s(nd,ns) -> k(nx,ns)
  The difference in speed is significant on the CPU but only around 50% on the GPU
*/

__global__ void _Ast_mul_Bs_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _As_mul_Bs_32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  float sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  double sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bs_32(int mx, int ns, float *x, float *sval, int *srow, int *scol, float *k) {
  int s0, s1, sp, sc, sr, x0, xr, k0, k1, kp;
  float sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s and k to be processed
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based), this value multiplies the sr'th column of x
      x0 = mx*sr;	// x[x0]: first element of x[:,sr]
      for (xr = 0; xr < mx; xr++) {
	xv = x[x0+xr];     // xv: x[xr,sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) {
  int s0, s1, sp, sc, sr, x0, xr, k0, k1, kp;
  double sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s and k to be processed
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based), this value multiplies the sr'th column of x
      x0 = mx*sr;	// x[x0]: first element of x[:,sr]
      for (xr = 0; xr < mx; xr++) {
	xv = x[x0+xr];     // xv: x[xr,sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}


/* We will do dw=dy*x' where x is a sparse matrix one column of x at a time. */

__global__ void _A_mul_Bst_32(int my, int xc, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
  // dw[wr,wc] += dy[yr,yc] * x[xr,xc]  where wr=yr, wc=xr, yc=xc
  int t, n, xp, xr, yp, yr, wp;
  t = threadIdx.x + blockIdx.x * blockDim.x;
  n = xcol[xc+1] - xcol[xc];
  while (t < n) {
    xp = xcol[xc] + t - 1;
    xr = xrow[xp] - 1;
    for (yr = 0; yr < my; yr++) {
      yp = yr + xc * my;
      wp = yr + xr * my;
      dw[wp] += dy[yp] * xval[xp];
    }
    t += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bst_64(int my, int xc, double *dy, double *xval, int *xrow, int *xcol, double *dw) {
  // dw[wr,wc] += dy[yr,yc] * x[xr,xc]  where wr=yr, wc=xr, yc=xc
  int t, n, xp, xr, yp, yr, wp;
  t = threadIdx.x + blockIdx.x * blockDim.x;
  n = xcol[xc+1] - xcol[xc];
  while (t < n) {
    xp = xcol[xc] + t - 1;
    xr = xrow[xp] - 1;
    for (yr = 0; yr < my; yr++) {
      yp = yr + xc * my;
      wp = yr + xr * my;
      dw[wp] += dy[yp] * xval[xp];
    }
    t += blockDim.x * gridDim.x;
  }
}

__global__ void _axpb32(int n, float a, float b, float *x) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    x[i] = a * x[i] + b;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _axpb64(int n, double a, double b, double *x) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    x[i] = a * x[i] + b;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _mul2_32(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    z[i] = y[i] * x[i];
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _mul2_64(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    z[i] = y[i] * x[i];
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {

  void A_mul_Bs_32(int mx, int ns,  float *x,  float *sval, int *srow, int *scol,  float *k) KCALL(_A_mul_Bs_32,mx,ns,x,sval,srow,scol,k);
  void A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) KCALL(_A_mul_Bs_64,mx,ns,x,sval,srow,scol,k);
  void Ast_mul_Bs_32(int nx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol,  float *k) KCALL(_Ast_mul_Bs_32,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_Ast_mul_Bs_64,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void As_mul_Bs_32(int mx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol,  float *k) KCALL(_As_mul_Bs_32,mx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_As_mul_Bs_64,mx,ns,xval,xrow,xcol,sval,srow,scol,k);

  void A_mul_Bst_32(int my, int ny, int mx, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
    // dy[my,ny] * x[mx,nx]' -> w[mw,nw]   where ny=nx, mw=my, nw=mx
    CUDA(hipMemset(dw, 0, my * mx * sizeof(float)));
    CUDA(hipDeviceSynchronize());
    for (int xc=0; xc<ny; xc++) {		// do one column of x at a time (row of x')
      KCALL(_A_mul_Bst_32,my,xc,dy,xval,xrow,xcol,dw);
      CUDA(hipDeviceSynchronize());
    }
  }

  void A_mul_Bst_64(int my, int ny, int mx, double *dy, double *xval, int *xrow, int *xcol, double *dw) {
    // dy[my,ny] * x[mx,nx]' -> w[mw,nw]   where ny=nx, mw=my, nw=mx
    CUDA(hipMemset(dw, 0, my * mx * sizeof(double)));
    CUDA(hipDeviceSynchronize());
    for (int xc=0; xc<ny; xc++) {		// do one column of x at a time (row of x')
      KCALL(_A_mul_Bst_64,my,xc,dy,xval,xrow,xcol,dw);
      CUDA(hipDeviceSynchronize());
    }
  }

  // To test the blk,thr parameters:
  // #define KCALL(f,...) {f<<<BLK,THR>>>(__VA_ARGS__); CUDA(hipGetLastError()); }
  void At_test(int blk,int thr,int nx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol, float *k) {_Ast_mul_Bs_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }
  void A_test(int blk,int thr,int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {_As_mul_Bs_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }


  void axpb32(int n, float a, float b, float *x) KCALL(_axpb32,n,a,b,x);
  void axpb64(int n, double a, double b, double *x) KCALL(_axpb64,n,a,b,x);

  void mul2_32(int n, float  *x, float  *y,  float *z) KCALL(_mul2_32,n,x,y,z);
  void mul2_64(int n, double *x, double *y, double *z) KCALL(_mul2_64,n,x,y,z);
}
