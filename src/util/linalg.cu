#include "hip/hip_runtime.h"
#include "../kunet.h"

/*
  The following functions multiply two sparse matrices into a dense matrix.
  The sparse matrices are in 1-based csc format.
  At_mul_B uses the transpose of the first arg and a simpler algorithm.
  x(nd,nx) s(nd,ns) -> k(nx,ns)
  A_mul_B uses the fast algorithm from the Julia sparse code.
  x(nx,nd) s(nd,ns) -> k(nx,ns)
  The difference in speed is significant on the CPU but only around 50% on the GPU
*/

__global__ void _At_mul_B_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _At_mul_B_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_B_32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  float sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_B_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  double sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

extern "C" {

  void At_mul_B_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) KCALL(_At_mul_B_32,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void At_mul_B_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_At_mul_B_64,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void A_mul_B_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) KCALL(_A_mul_B_32,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void A_mul_B_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_A_mul_B_64,nx,ns,xval,xrow,xcol,sval,srow,scol,k);

  // To test the blk,thr parameters:
  // #define KCALL(f,...) {f<<<BLK,THR>>>(__VA_ARGS__); CUDA(hipGetLastError()); }
  void At_test(int blk,int thr,int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {_At_mul_B_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }
  void A_test(int blk,int thr,int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {_A_mul_B_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }

}