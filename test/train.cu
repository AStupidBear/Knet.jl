#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <assert.h>
#include "jnet.h"
#include "jnet_h5.h"

static clock_t t0;
#define tic (t0 = clock())
#define toc fprintf(stderr, "%g seconds\n", (double)(clock()-t0)/CLOCKS_PER_SEC)

const char *usage =
  "Usage: %s [opts] x layer1 layer2 ... y\n"  
  "where each of x layer1 ... y is an hdf5 file\n"
  "-b batchsize (default: size of dataset)\n"
  "-o prefix (default: train.out)\n"
  "   the new layers will be saved in prefix1.h5, prefix2.h5, ... etc.\n"
  "-a (adagrad, default:false)\n"
  "-i iters (default: until one epoch is completed)\n";

int main(int argc, char **argv) {
  int batch = 0;
  int adagrad = 0;
  int iters = 0;
  const char *output = "train.out";
  int opt;
  while((opt = getopt(argc, argv, "o:b:i:a")) != -1) {
    switch(opt) {
    case 'b': batch = atoi(optarg); break;
    case 'i': iters = atoi(optarg); break;
    case 'o': output = optarg; break;
    case 'a': adagrad = 1; break;
    default: fprintf(stderr, usage, argv[0]); exit(EXIT_FAILURE);
    }
  }
  if (argc - optind < 3) {
    fprintf(stderr, usage, argv[0]); exit(EXIT_FAILURE);
  }

  float *x; int xrows, xcols;
  fprintf(stderr, "Reading %s... ", argv[optind]);
  tic; h5read(argv[optind++], &xrows, &xcols, &x); toc;
  
  int nlayers = argc - optind - 1;
  Layer *net = (Layer *) calloc(nlayers, sizeof(Layer));
  fprintf(stderr, "Reading "); tic;
  for (int l = 0; l < nlayers; l++) {
    fprintf(stderr, "%s... ", argv[optind + l]);
    net[l] = h5read_layer(argv[optind + l]);
    if (adagrad) net[l]->adagrad = 1;
  }
  optind += nlayers;
  toc;

  float *y; int yrows, ycols;
  fprintf(stderr, "Reading %s... ", argv[optind]);
  tic; h5read(argv[optind++], &yrows, &ycols, &y); toc;
  assert(yrows == net[nlayers-1]->wrows);
  assert(ycols == xcols);

  if (batch == 0) batch = xcols;
  if (iters > 0) { 
    assert(xcols >= batch * iters); 
    fprintf(stderr, "Training %d iters of %d batches... ", iters, batch);
    xcols = batch * iters; 
  } else {
    fprintf(stderr, "Training a single epoch with batch=%d... ", batch);
  }
  tic; train(net, x, y, nlayers, xcols, batch); toc;

  char *fname = (char *) malloc(strlen(output) + 128);
  fprintf(stderr, "Saving resulting layers "); tic;
  for (int l = 0; l < nlayers; l++) {
    sprintf(fname, "%s%d.h5", output, l+1);
    fprintf(stderr, "%s... ", fname);
    h5write_layer(fname, net[l]);
  }
  toc;
  free(fname);

  for (int l = 0; l < nlayers; l++) lfree(net[l]);
  free(x); free(y); free(net);
}
